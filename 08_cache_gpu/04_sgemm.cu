
#include <hip/hip_runtime.h>
#include <cmath>
#include <hipblas.h>
#include <cstdlib>
#include <cstdio>
#include <chrono>
using namespace std;

#define M 1024

int main(int argc, char **argv) {
  int N = 2048;
  int Nt = 10;
  int size = N * N * sizeof(float);
  float *A, *B, *C;
  hipMallocManaged(&A, size);
  hipMallocManaged(&B, size);
  hipMallocManaged(&C, size);
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      A[N*i+j] = drand48();
      B[N*i+j] = drand48();
      C[N*i+j] = 0;
    }
  }
  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  auto tic = chrono::steady_clock::now();
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
		&alpha, B, N, A, N, &beta, C, N);
  }
  hipDeviceSynchronize();
  auto toc = chrono::steady_clock::now();
  hipblasGetMatrix(N, N, sizeof(*C), C, N, C, N);
  double time = chrono::duration<double>(toc - tic).count() / Nt;
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
#pragma omp parallel for
  for (int i=0; i<N; i++)
    for (int k=0; k<N; k++)
      for (int j=0; j<N; j++)
        C[N*i+j] -= A[N*i+k] * B[N*k+j];
  double err = 0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      err += fabs(C[N*i+j]);
  printf("error: %lf\n",err/N/N);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipblasDestroy(handle);
}
