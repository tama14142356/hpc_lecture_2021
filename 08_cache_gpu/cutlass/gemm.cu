#include "hip/hip_runtime.h"
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <hipblas.h>
#define DEBUG

#include "cutlass.h"
#include "timer.h"
using namespace cutlass;

int main(int argc, const char **argv) {
  int m = 10240;
  int k = 4096;
  int n = 8192;
  float alpha = 1.0;
  float beta = 0.0;
  typedef float value_t;
  typedef float accum_t;
  int g_timing_iterations = 10;
  float *A, *B, *C, *C2;
  hipMallocManaged(&A, m * k * sizeof(float));
  hipMallocManaged(&B, k * n * sizeof(float));
  hipMallocManaged(&C, m * n * sizeof(float));
  hipMallocManaged(&C2, m * n * sizeof(float));
  for (int i=0; i<m; i++)
    for (int j=0; j<k; j++)
      A[k*i+j] = drand48();
  for (int i=0; i<k; i++)
    for (int j=0; j<n; j++)
      B[n*i+j] = drand48();
  for (int i=0; i<n; i++) {
    for (int j=0; j<m; j++) {
      C[m*i+j] = drand48();
      C2[m*i+j] = drand48();
    }
  }
  hipblasHandle_t g_cublas_handle;
  hipblasCreate(&g_cublas_handle);
  gpu_timer timer;
  for (int i = 0; i < g_timing_iterations+2; i++) {
    if (i == 2) timer.start();
    hipblasSgemm(g_cublas_handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		m,
		n,
		k,
		&alpha,
		A,
		m,
		B,
		k,
		&beta,
		C,
		m);
  }
  timer.stop();
  int64_t num_flops = (2 * int64_t(m) * int64_t(n) * int64_t(k)) + (2 * int64_t(m) * int64_t(n));
  double tcublas = timer.elapsed_millis() / g_timing_iterations;
  double cublas_flops = double(num_flops) / tcublas / 1.0e6;
  int tile = 64;
  dim3 block = dim3(tile);
  dim3 grid = dim3((m+tile-1)/tile, (n+tile-1)/tile);
  for (int i = 0; i < g_timing_iterations+2; i++) {
    if (i == 2) timer.start();
    kernel<<< grid, block >>>(m,
			      n,
			      k,
			      A,
			      B,
			      C2);
  }
  timer.stop();
  double tcutlass = timer.elapsed_millis() / g_timing_iterations;
  double cutlass_flops = double(num_flops) / tcutlass / 1.0e6;
  printf("CUBLAS: %.2f Gflops, CUTLASS: %.2f Gflops\n", cublas_flops, cutlass_flops);
  double err = 0;
  for (int i=0; i<n; i++) {
    for (int j=0; j<m; j++) {
      err += fabs(C[m*i+j] - C2[m*i+j]);
    }
  }
  printf("error: %lf\n", err/n/m);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(C2);
  hipblasDestroy(g_cublas_handle);
}
