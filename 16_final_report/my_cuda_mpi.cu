#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>
#include <cmath>
#include <vector>
#include <chrono>
using namespace std;

__global__ void matmul(float *A, float *B, float *C, int N, int offset) {
  int i = blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  int n = blockDim.x;
  float sum = 0.0f;
  extern __shared__ float A_s[];
  for (int ks=0; ks<N; ks+=blockDim.x) {
    __syncthreads();
    A_s[threadIdx.x] = A[N*i+ks+threadIdx.x];
    __syncthreads();
    for (int k=ks; k<ks+blockDim.x; k++) {
      sum += A_s[k-ks] * B[n*k+j];
    }
  }
  C[N*i+j+offset] = sum;
}

int main(int argc, char** argv) {
  char hostname[256];
  int size, rank;
  int gpusize, gpurank, len;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Get_processor_name(hostname, &len);
  hipGetDeviceCount(&gpusize);
  hipSetDevice(rank % gpusize);
  hipGetDevice(&gpurank);

  const int N = 2048;
  const int M = min(1024, N/size);
  int mat_size = N * N * sizeof(float);
  int sub_mat_size = N * N * sizeof(float);
  float *A, *B, *C, *subA, *subB, *subC, *recv;
  hipMallocManaged(&A, mat_size);
  hipMallocManaged(&B, mat_size);
  hipMallocManaged(&C, mat_size);
  hipMallocManaged(&subA, sub_mat_size);
  hipMallocManaged(&subB, sub_mat_size);
  hipMallocManaged(&subC, sub_mat_size);
  hipMallocManaged(&recv, sub_mat_size);
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      A[N*i+j] = drand48();
      B[N*i+j] = drand48();
    }
  }
  int offset = N/size*rank;
  for (int i=0; i<N/size; i++)
    for (int j=0; j<N; j++)
      subA[N*i+j] = A[N*(i+offset)+j];
  for (int i=0; i<N; i++)
    for (int j=0; j<N/size; j++)
      subB[N/size*i+j] = B[N*i+j+offset];
  int recv_from = (rank + 1) % size;
  int send_to = (rank - 1 + size) % size;

  double comp_time = 0, comm_time = 0;
  for(int irank=0; irank<size; irank++) {
    auto tic = chrono::steady_clock::now();
    offset = N/size*((rank+irank) % size);
    dim3 grid(N/(size*M), N);
    matmul<<<grid,M,M*sizeof(float)>>>(subA, subB, subC, N, offset);
    auto toc = chrono::steady_clock::now();
    hipDeviceSynchronize();
    comp_time += chrono::duration<double>(toc - tic).count();
    MPI_Request request[2];
    MPI_Isend(&subB[0], N*N/size, MPI_FLOAT, send_to, 0, MPI_COMM_WORLD, &request[0]);
    MPI_Irecv(&recv[0], N*N/size, MPI_FLOAT, recv_from, 0, MPI_COMM_WORLD, &request[1]);
    MPI_Waitall(2, request, MPI_STATUS_IGNORE);
    for (int i=0; i<N*N/size; i++)
      subB[i] = recv[i];
    tic = chrono::steady_clock::now();
    comm_time += chrono::duration<double>(tic - toc).count();
  }
  MPI_Allgather(&subC[0], N*N/size, MPI_FLOAT, &C[0], N*N/size, MPI_FLOAT, MPI_COMM_WORLD);
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      for (int k=0; k<N; k++)
        C[N*i+j] -= A[N*i+k] * B[N*k+j];
  double err = 0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      err += fabs(C[N*i+j]);
  if(rank==0) {
    double time = comp_time+comm_time;
    printf("N    : %d\n",N);
    printf("comp : %lf s\n", comp_time);
    printf("comm : %lf s\n", comm_time);
    printf("total: %lf s (%lf GFlops)\n",time,2.*N*N*N/time/1e9);
    printf("error: %lf\n",err/N/N);
  }
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(subA);
  hipFree(subB);
  hipFree(subC);
  hipFree(recv);
  MPI_Finalize();
}
